#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>

struct point
{
	float x;
	float y;
};	
struct dist
{
	float da;
	float db;
	float dc;
};

float eucli(float fx, float fy)
{
	return sqrt(fx * fx + fy * fy);
}

__global__ void trilaterate(struct point a, struct point b, struct point c, struct dist *d_set, struct point *d_trail, int NUM)
{
	float a1Sq = a.x * a.x, a2Sq = b.x * b.x, a3Sq = c.x * c.x, b1Sq = a.y * a.y, b2Sq = b.y * b.y, b3Sq = c.y * c.y;
	float r1Sq, r2Sq, r3Sq, denom1, numer1, denom2, numer2;
	float a1 = a.x, a2 = b.x, a3 = c.x, b1 = a.y, b2 = b.y, b3 = c.y; 
	int i;
	
	for(i=0; i < NUM; i++)
	{
			r1Sq = d_set[i].da * d_set[i].da;
			r2Sq = d_set[i].db * d_set[i].db;
			r3Sq = d_set[i].dc * d_set[i].dc;
			
			numer1 = (a2 - a1) * (a3Sq + b3Sq - r3Sq) + (a1 - a3) * (a2Sq + b2Sq - r2Sq) + (a3 - a2) * (a1Sq + b1Sq - r1Sq);
			denom1 = 2 * (b3 * (a2 - a1) + b2 * (a1 - a3) + b1 * (a3 - a2));
			d_trail[i].y = numer1/denom1;
			
			numer2 = r2Sq - r1Sq + a1Sq - a2Sq + b1Sq - b2Sq - 2 * (b1 - b2) * d_trail[i].y;
			denom2 = 2 * (a1 - a2);
			d_trail[i].x = numer2/denom2;
	}
}

int main(int argc, char *argv[])
{
	hipEvent_t start, stop;
	float etime;
	int i, j=0;
	float fx, fy, gx, gy, z = 5.0;
	
	int NUM;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	if (argc != 2) 
	{
	  printf("Check you arguments!\n");
	  exit(1);
    }
	
	struct point a, b, c;
	
	a.x = 1.67; a.y = 2.58;
	b.x = 3.74; b.y = 2.08;
	c.x = 5.12; c.y = 3.95;
	
	struct point init;
	init.x = 3.12;
	init.y = 4.27;
	
	NUM = atoi(argv[1]);
	
	struct point trail[NUM], avg_trail[(NUM/4)], ret_avg_trail[(NUM/4)];
	struct point *d_trail, *h_trail;
	
	trail[0] = init;
	
	srand(time(NULL));
	
	for(i=1; i<NUM; i++)
	{
		gx = ((float)rand()/(float)(RAND_MAX)) * z;
		gx = floorf(gx * 100) / 100;
		gy = ((float)rand()/(float)(RAND_MAX)) * z;
		gy = floorf(gy * 100) / 100;
		trail[i].x = (floorf(trail[i-1].x * 100 + 0.5) / 100) + gx;
		trail[i].y = (floorf(trail[i-1].y * 100 + 0.5) / 100) + gy;	
	}
	
	for(i=0; i<(NUM/4); i++)
	{
		avg_trail[i].x = (trail[j].x + trail[j+1].x + trail[j+2].x + trail[j+3].x) / 4;
		avg_trail[i].y = (trail[j].y + trail[j+1].y + trail[j+2].y + trail[j+3].y) / 4;
		j += 4;
	}
	
	printf("\nAvg. Random Trail at Host\n");
	for(i=0; i<(NUM/4); i++)
	{
		printf("(%f, %f)\n", avg_trail[i].x, avg_trail[i].y);
	}
	
	struct dist *set;
	
	size_t size = NUM * sizeof(struct dist);
	set = (struct dist *)malloc(size);
	
	size_t sz = NUM * sizeof(struct point);
	h_trail = (struct point *)malloc(sz);
	
	for(i=0; i<NUM; i++)
	{
		fx = trail[i].x - a.x;
		fy = trail[i].y - a.y;
		set[i].da = eucli(fx, fy);
		fx = trail[i].x - b.x;
		fy = trail[i].y - b.y;
		set[i].db = eucli(fx, fy);
		fx = trail[i].x - c.x;
		fy = trail[i].y - c.y;
		set[i].dc = eucli(fx, fy);
	}
	
	struct dist *d_set;
	hipMalloc((void **) &d_set, size);
	
	hipMalloc((void **) &d_trail, sz);
	
	hipMemcpy(d_set, set, sizeof(struct dist)*NUM, hipMemcpyHostToDevice);
	
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
    int nBlocks = devProp.multiProcessorCount;
	int blockSize = devProp.warpSize;	

	printf("\nU: %d\n", nBlocks);
	printf("\nV: %d\n", blockSize);

	trilaterate <<< nBlocks, blockSize >>> (a, b, c, d_set, d_trail, NUM);
	
	hipMemcpy(h_trail, d_trail, sizeof(struct point)*NUM, hipMemcpyDeviceToHost);
	
	j=0;
	for(i=0; i<(NUM/4); i++)
	{
		ret_avg_trail[i].x = (h_trail[j].x + h_trail[j+1].x + h_trail[j+2].x + h_trail[j+3].x) / 4;
		ret_avg_trail[i].y = (h_trail[j].y + h_trail[j+1].y + h_trail[j+2].y + h_trail[j+3].y) / 4;
		j += 4;
	}
	
	printf("\nAvg. Generated Trail at Device\n");
	for(i=0; i<(NUM/4); i++)
	{
		printf("(%f, %f)\n", ret_avg_trail[i].x, ret_avg_trail[i].y);
	}
	printf("\n");
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&etime, start, stop);
	printf("Time elapsed: %f ms\n", etime);

    hipEventDestroy(start);
	hipEventDestroy(stop);
	free(set); 
	hipFree(d_set);
	hipFree(d_trail);
	hipFree(h_trail);
	
	return 0;
}
